
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void init(int *key) {
  key[threadIdx.x] = 0;
}

__global__ void scan(int *a, int *b, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for(int j=1; j<N; j<<=1) {
    b[i] = a[i];
    __syncthreads();
    a[i] += b[i-j];
    __syncthreads();
  }
}

__global__ void bucketCount(int *bucket, int *key) {
  atomicAdd(&bucket[key[threadIdx.x]], 1);
}

__global__ void sort(int *key, int *bucket, int n, int range) {
  int begin = threadIdx.x == 0 ? 0 : bucket[threadIdx.x-1];
  int end = bucket[threadIdx.x];
  if (blockIdx.x >= end || blockIdx.x < begin) return;
  key[blockIdx.x] = threadIdx.x;
}

int main() {
  int n = 50;
  int range = 5;

  int *key, *bucket, *tmp;
  hipMallocManaged(&key, n*sizeof(int));
  hipMallocManaged(&bucket, range*sizeof(int));
  hipMallocManaged(&tmp, range*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  init<<<1,range>>>(bucket);
  hipDeviceSynchronize();

  bucketCount<<<1,n>>>(bucket, key);
  hipDeviceSynchronize();

  scan<<<1,range>>>(bucket, tmp, range);
  hipDeviceSynchronize();

  sort<<<n,range>>>(key, bucket, n, range);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");

  hipFree(key);
  hipFree(bucket);
  hipFree(tmp);
}

